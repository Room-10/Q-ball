#include "hip/hip_runtime.h"

__global__ void linop1(double *x, double *ygrad)
{
    /* ggrad^ij = A^j' w^ij
     *
     * pgrad = 0
     * pgrad_t^i += - P^j' B^j' w_t^ij
     *
     * q0grad = b'u1
     */

    SUBVAR_x_u1(u1,x)
    SUBVAR_x_w(w,x)
    SUBVAR_y_q0(q0grad,ygrad)
    SUBVAR_y_g(ggrad,ygrad)
    SUBVAR_y_p(pgrad,ygrad)

    // global thread index
    int _mj = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int t = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if (_mj >= sm_skip || i >= n_image || t >= d_image)
       return;

    // these have to be computed on the fly
    int m = _mj / m_gradients;
    int j = _mj % m_gradients;

    // iteration variables and misc.
    int ll, mm, jj, kk, idx;
    double newval;

    // ggrad[i,j,m,t]
    idx = i*msd_skip + j*sd_skip + m*d_image + t;
    newval = 0.0;

    // ggrad^ij = A^j' w^ij
    for (ll = 0; ll < s_manifold; ll++) {
        // jlm,ijlt->ijmt
        newval += A[j*ss_skip + ll*s_manifold + m] *
                    w[i*msd_skip + j*sd_skip + ll*d_image + t];
    }
    ggrad[idx] = newval;

    if (_mj == 0) {
        // pgrad = 0
        for (kk = 0; kk < l_labels; kk++) {
            pgrad[kk*nd_skip + t*n_image + i] = 0.0;
        }

        // pgrad_t^i += - P^j' B^j' w_t^ij
        for (jj = 0; jj < m_gradients; jj++) {
            for (mm = 0; mm < r_points; mm++) {
                idx = P[jj*r_points + mm]*nd_skip + t*n_image + i;
                newval = pgrad[idx];
                for (ll = 0; ll < s_manifold; ll++) {
                    // jlm,ijlt->jmti
                    newval -= B[jj*sr_skip + ll*r_points + mm] *
                                w[i*msd_skip + jj*sd_skip + ll*d_image + t];
                }
                pgrad[idx] = newval;
            }
        }

        if (t == 0) {
            // q0grad = b'u1
            newval = 0.0;
            for (kk = 0; kk < l_labels; kk++) {
                newval += b[kk]*u1[kk*n_image + i];
            }
            newval *= b_precond;
            q0grad[i] = newval;
        }
    }
}

__global__ void linop2(double *x, double *ygrad)
{
    /* pgrad += diag(b) D u1 (D is the gradient on a staggered grid)
     *
     * q1grad = Y v - u1
     *
     * q2grad = Y M v - u2
     */

    SUBVAR_x_u1(u1,x)
    SUBVAR_x_u2(u2,x)
    SUBVAR_x_v(v,x)
    SUBVAR_y_p(pgrad,ygrad)
    SUBVAR_y_q1(q1grad,ygrad)
    SUBVAR_y_q2(q2grad,ygrad)

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int t = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if (k >= l_labels || i >= n_image || t >= d_image)
       return;

    // iteration variable and misc.
    int aa, base, idx;
    double newval, fac;

    // skip points on "bottom right" boundary
    int is_boundary = false;
    int curr_i = i, curr_dim = 0;
    for (aa = d_image - 1; aa >= 0; aa--) {
        curr_dim = curr_i / skips[aa];
        curr_i = curr_i % skips[aa];
        if (curr_dim == imagedims[d_image - 1 - aa] - 1) {
            is_boundary = true;
            break;
        }
    }
    // pgrad[k,t,i]
    idx = k*nd_skip + t*n_image + i;
    newval = pgrad[idx];
    fac = b[k]/(double)navgskips;

    // pgrad += diag(b) D u1 (D is the gradient on a staggered grid)
    if (!is_boundary) {
        for (aa = 0; aa < navgskips; aa++) {
            base = i + avgskips[t*navgskips + aa];
            newval += fac*(
                u1[k*n_image + (base + skips[t])] - u1[k*n_image + base]
            );
        }
    }
    pgrad[idx] = newval;

    if (t == 0) {
        // q1grad[k,i]
        idx = k*n_image + i;
        newval = -u1[idx];

        // q1grad = Y v - u1
        for (aa = 0; aa < l_shm; aa++) {
            newval += Y[k*l_shm + aa]*v[aa*n_image + i];
        }
        q1grad[idx] = newval;

        // q2grad[k,i]
        idx = k*n_image + i;
        newval = -u2[idx];

        // q2grad = Y M v - u2
        for (aa = 0; aa < l_shm; aa++) {
            newval += Y[k*l_shm + aa]*M[aa]*v[aa*n_image + i];
        }
        q2grad[idx] = newval;
    }
}

#ifdef precond
__global__ void prox_dual(double *y, double *ysigma)
#else
__global__ void prox_dual(double *y, double sigma)
#endif
{
    /* This function makes heavy use of registers (34 32-bit registers), so
     * that it will not run with more than 960 threads per block on compute
     * capability 2.x!
     *
     * g = proj(g, lbd)
     * q0 -= sigma
     */

    SUBVAR_y_g(g,y)
    SUBVAR_y_q0(q0,y)

#if (d_image <= s_manifold)
// A := gij, a (d_image x s_manifold)-matrix
#define LIM s_manifold
#define STEP1 d_image
#define STEP2 (1)
#else
// A := gij^T, a (s_manifold x d_image)-matrix
#define LIM d_image
#define STEP1 (1)
#define STEP2 d_image
#endif

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if (i >= n_image || j >= m_gradients)
       return;

    // iteration variables and misc.
    int mm;
    double *gij = &g[i*msd_skip + j*sd_skip];
    double norm = 0.0;

    // g = proj(g, lbd)
#if (d_image == 1 || s_manifold == 1)
    for (mm = 0; mm < LIM; mm++) {
        norm += gij[mm*STEP1]*gij[mm*STEP1];
    }

    if (norm > lbd*lbd) {
        norm = lbd/sqrt(norm);
        for (mm = 0; mm < LIM; mm++) {
            gij[mm*STEP1] *= norm;
        }
    }
#elif (d_image == 2 || s_manifold == 2)
    double C11 = 0.0, C12 = 0.0, C22 = 0.0,
           V11 = 0.0, V12 = 0.0, V21 = 0.0, V22 = 0.0,
           M11 = 0.0, M12 = 0.0, M21 = 0.0, M22 = 0.0,
           s1 = 0.0, s2 = 0.0,
           trace, d, lmax, lmin, smax, smin;

    // C = A^T A, a (2 x 2)-matrix
    for (mm = 0; mm < LIM; mm++) {
        C11 += gij[mm*STEP1 + 0*STEP2]*gij[mm*STEP1 + 0*STEP2];
        C12 += gij[mm*STEP1 + 0*STEP2]*gij[mm*STEP1 + 1*STEP2];
        C22 += gij[mm*STEP1 + 1*STEP2]*gij[mm*STEP1 + 1*STEP2];
    }

    // Compute eigenvalues
    trace = C11 + C22;
    d = sqrt(fmax(0.0, 0.25*trace*trace - (C11*C22 - C12*C12)));
    lmax = fmax(0.0, 0.5*trace + d);
    lmin = fmax(0.0, 0.5*trace - d);
    smax = sqrt(lmax);
    smin = sqrt(lmin);

    if (smax > lbd) {
        // Compute orthonormal eigenvectors
        if (C12 == 0.0) {
            if (C11 >= C22) {
                V11 = 1.0; V12 = 0.0;
                V21 = 0.0; V22 = 1.0;
            } else {
                V11 = 0.0; V12 = 1.0;
                V21 = 1.0; V22 = 0.0;
            }
        } else {
            V11 = C12       ; V12 = C12;
            V21 = lmax - C11; V22 = lmin - C11;
            norm = hypot(V11, V21);
            V11 /= norm; V21 /= norm;
            norm = hypot(V12, V22);
            V12 /= norm; V22 /= norm;
        }

        // Thresholding of eigenvalues
        s1 = fmin(smax, lbd)/smax;
        s2 = fmin(smin, lbd);
        s2 = (smin > 0.0) ? s2/smin : 0.0;

        // M = V * diag(s) * V^T
        M11 = s1*V11*V11 + s2*V12*V12;
        M12 = s1*V11*V21 + s2*V12*V22;
        M21 = s1*V21*V11 + s2*V22*V12;
        M22 = s1*V21*V21 + s2*V22*V22;

        // proj(A) = A * M
        for (mm = 0; mm < LIM; mm++) {
            // s1, s2 now used as temp. variables
            s1 = gij[mm*STEP1 + 0*STEP2];
            s2 = gij[mm*STEP1 + 1*STEP2];
            gij[mm*STEP1 + 0*STEP2] = s1*M11 + s2*M21;
            gij[mm*STEP1 + 1*STEP2] = s1*M12 + s2*M22;
        }
    }
#endif

    if (j == 0) {
#ifdef precond
        SUBVAR_y_q0(q0sigma,ysigma)
        q0[i] -= q0sigma[i]*b_precond;
#else
        q0[i] -= sigma*b_precond;
#endif
    }
}
