#include "hip/hip_runtime.h"

__global__ void PrimalKernel1(KERNEL_PARAMS)
{
    /* ubark = 0
     * ubark += diag(b) D' pkp1 (D' = -div with Dirichlet boundary)
     */

    SUBVAR_ubark
    SUBVAR_pkp1

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;

    // stay inside maximum dimensions
    if(k >= l_labels)
       return;

    // iteration variables and misc.
    int ii, aa, tt, is_boundary, curr_dim, curr_i, base;

    for(ii = 0; ii < n_image; ii++) {
        ubark[k*n_image + ii] = 0.0;
    }

    for(ii = 0; ii < n_image; ii++) {
        // skip points on "bottom right" boundary
        is_boundary = false; curr_i = ii;
        for(tt = d_image - 1; tt >= 0; tt--) {
            curr_dim = curr_i / skips[tt];
            curr_i = curr_i % skips[tt];
            if(curr_dim == imagedims[d_image - 1 - tt] - 1) {
                is_boundary = true;
                break;
            }
        }

        if(!is_boundary) {
            for(tt = 0; tt < d_image; tt++) {
                for(aa = 0; aa < navgskips; aa++) {
                    base = ii + avgskips[tt*navgskips + aa];
                    ubark[k*n_image + (base + skips[tt])] +=
                        b[k]/(double)navgskips * pkp1[k*nd_skip + tt*n_image + ii];
                    ubark[k*n_image + base] -=
                        b[k]/(double)navgskips * pkp1[k*nd_skip + tt*n_image + ii];
                }
            }
        }
    }
}

__global__ void PrimalKernel2(KERNEL_PARAMS)
{
    /* wbark^ij = A^j gkp1_t^ij - B^j P^j pkp1_t^i
     * wbark = wk - tau*wbark
     * wbark, wk = wbark + theta*(wbark - wk), wbark
     *
     * (W1:)
     * w0bark^ij = A^j g0kp1_t^ij - B^j P^j p0kp1_t^i
     * w0bark = w0k - tau*w0bark
     * w0bark, w0k = w0bark + theta*(w0bark - w0k), w0bark
     */

    SUBVAR_wk
    SUBVAR_wbark
    SUBVAR_wkp1
    SUBVAR_w0k
    SUBVAR_w0bark
    SUBVAR_w0kp1
    SUBVAR_pkp1
    SUBVAR_gkp1
    SUBVAR_p0kp1
    SUBVAR_g0kp1

    // global thread index
    int _lj = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int t = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if(_lj >= sm_skip || i >= n_image || t >= d_image)
       return;

    // these have to be computed on the fly
    int l = _lj / m_gradients;
    int j = _lj % m_gradients;

    // iteration variable and misc.
    int mm;
    double wkp1_tmp;

    wkp1_tmp = 0.0;
    for(mm = 0; mm < s_manifold; mm++) {
        // jlm,ijmt->ijlt
        wkp1_tmp += A[j*ss_skip + l*s_manifold + mm] *
                    gkp1[i*msd_skip + j*sd_skip + mm*d_image + t];
    }

    for(mm = 0; mm < r_points; mm++) {
        // jlm,jmti->ijlt
        wkp1_tmp -= B[j*sr_skip + l*r_points + mm] *
                    pkp1[P[j*r_points + mm]*nd_skip + t*n_image + i];
    }

    wkp1_tmp = wk[i*msd_skip + j*sd_skip + l*d_image + t] - tau*wkp1_tmp;

    wbark[i*msd_skip + j*sd_skip + l*d_image + t] =
        (1 + theta)*wkp1_tmp - theta*wk[i*msd_skip + j*sd_skip + l*d_image + t];
    wk[i*msd_skip + j*sd_skip + l*d_image + t] = wkp1_tmp;
    wkp1[i*msd_skip + j*sd_skip + l*d_image + t] = wkp1_tmp;

#if 'W' == dataterm
    if(t == 0) {
        wkp1_tmp = 0.0;
        for(mm = 0; mm < s_manifold; mm++) {
            // jlm,ijm->ijl
            wkp1_tmp += A[j*ss_skip + l*s_manifold + mm] *
                        g0kp1[i*sm_skip + j*s_manifold + mm];
        }
        for(mm = 0; mm < r_points; mm++) {
            // jlm,jmi->ijl
            wkp1_tmp -= B[j*sr_skip + l*r_points + mm] *
                        p0kp1[P[j*r_points + mm]*n_image + i];
        }
        wkp1_tmp = w0k[i*sm_skip + j*s_manifold + l] - tau*wkp1_tmp;

        w0bark[i*sm_skip + j*s_manifold + l] =
            (1 + theta)*wkp1_tmp - theta*w0k[i*sm_skip + j*s_manifold + l];
        w0k[i*sm_skip + j*s_manifold + l] = wkp1_tmp;
        w0kp1[i*sm_skip + j*s_manifold + l] = wkp1_tmp;
    }
#endif
}

__global__ void PrimalKernel3(KERNEL_PARAMS)
{
    /* ubark += b q0kp1' - q1kp1 - diag(b) f (quadratic)
     * ubark += b q0kp1' - q1kp1 + diag(b) p0kp1 (W1)
     * ubark = dataterm_factor*(uk - tau*ubark)
     * ubark[~uconstrloc] = max(0, ubark)
     * ubark[uconstrloc] = constraint_u[uconstrloc]
     * ubark, uk = ubark + theta*(ubark - uk), ubark
     */

    SUBVAR_uk
    SUBVAR_ubark
    SUBVAR_ukp1
    SUBVAR_q0kp1
    SUBVAR_q1kp1
    SUBVAR_p0kp1

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(i >= n_image || k >= l_labels)
       return;

    // misc.
    double ukp1_tmp;

    double dataterm_factor = 1.0;
#if 'Q' == dataterm
    dataterm_factor = 1.0/(1.0 + tau*b[k]);
#endif

    if(uconstrloc[i]) {
        ukp1_tmp = constraint_u[k*n_image + i];
    } else {
        ukp1_tmp = ubark[k*n_image + i];
        ukp1_tmp += b[k]*(b_precond*q0kp1[i]);
        ukp1_tmp -= q1kp1[k*n_image + i];
#if 'Q' == dataterm
        ukp1_tmp -= b[k]*f[k*n_image + i];
#elif 'W' == dataterm
        ukp1_tmp += b[k]*p0kp1[k*n_image + i];
#endif
        ukp1_tmp = dataterm_factor*(uk[k*n_image + i] - tau*ukp1_tmp);
        ukp1_tmp = fmax(0.0,  ukp1_tmp);
    }

    ubark[k*n_image + i] = (1 + theta)*ukp1_tmp - theta*uk[k*n_image + i];
    uk[k*n_image + i] = ukp1_tmp;
    ukp1[k*n_image + i] = ukp1_tmp;
}

__global__ void PrimalKernel4(KERNEL_PARAMS)
{
    /* vbark^i = Y' q1kp1^i
     * vbark = vk - tau*vbark
     * vbark, vk = vbark + theta*(vbark - vk), vbark
     */

    SUBVAR_vk
    SUBVAR_vbark
    SUBVAR_vkp1
    SUBVAR_q1kp1

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int m = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(i >= n_image || m >= l_shm)
       return;

    // iteration variable and misc.
    int k;
    double vkp1_tmp;

    vkp1_tmp = 0.0;
    for(k = 0; k < l_labels; k++) {
        // km,ki->mi
        vkp1_tmp += Y[k*l_shm + m]*q1kp1[k*n_image + i];
    }

    vkp1_tmp = vk[m*n_image + i] - tau*vkp1_tmp;

    vbark[m*n_image + i] = (1 + theta)*vkp1_tmp - theta*vk[m*n_image + i];
    vk[m*n_image + i] = vkp1_tmp;
    vkp1[m*n_image + i] = vkp1_tmp;
}
