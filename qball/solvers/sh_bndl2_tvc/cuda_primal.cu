#include "hip/hip_runtime.h"

__global__ void linop_adjoint1(double *xgrad, double *y)
{
    /* vgrad = 0
     * vgrad += D' p (D' = -div with Dirichlet boundary)
     */

    SUBVAR_x_v(vgrad,xgrad)
    SUBVAR_y_p(p,y)

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;

    // stay inside maximum dimensions
    if(k >= l_shm)
       return;

    // iteration variables and misc.
    int ii, aa, tt, is_boundary, curr_dim, curr_i, base;
    double fac = 1.0/(double)navgskips;

    for(ii = 0; ii < n_image; ii++) {
        vgrad[k*n_image + ii] = 0.0;
    }

    // vgrad += diag(b) D' p (D' = -div with Dirichlet boundary)
    for(ii = 0; ii < n_image; ii++) {
        // skip points on "bottom right" boundary
        is_boundary = false; curr_i = ii;
        for(tt = d_image - 1; tt >= 0; tt--) {
            curr_dim = curr_i / skips[tt];
            curr_i = curr_i % skips[tt];
            if(curr_dim == imagedims[d_image - 1 - tt] - 1) {
                is_boundary = true;
                break;
            }
        }

        if(!is_boundary) {
            for(tt = 0; tt < d_image; tt++) {
                for(aa = 0; aa < navgskips; aa++) {
                    base = ii + avgskips[tt*navgskips + aa];
                    vgrad[k*n_image + (base + skips[tt])] +=
                        fac*p[k*nd_skip + tt*n_image + ii];
                    vgrad[k*n_image + base] -=
                        fac*p[k*nd_skip + tt*n_image + ii];
                }
            }
        }
    }
}

__global__ void linop_adjoint2(double *xgrad, double *y)
{
    /* u1grad = b q0' - q1
     * u2grad = -q2
     *
     * vgrad^i += Y'q1^i + M Y'q2^i
     */

    SUBVAR_x_u1(u1grad,xgrad)
    SUBVAR_x_u2(u2grad,xgrad)
    SUBVAR_x_v(vgrad,xgrad)
    SUBVAR_y_q0(q0,y)
    SUBVAR_y_q1(q1,y)
    SUBVAR_y_q2(q2,y)

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y;
    int m = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if (i >= n_image || k >= l_labels || m >= l_shm)
       return;

    // misc.
    int kk, idx;
    double newval;

    if (m == 0) {
        // u1grad[k,i]
        idx = k*n_image + i;

        // u1grad = b q0' - q1
        u1grad[idx] = b[k]*(b_precond*q0[i]) - q1[idx];

        if (inpaint_nloc[i]) {
            // u2grad = -q2
            u2grad[idx] = -q2[idx];
        }
    }

    if (k == 0) {
        // vgrad[m,i]
        idx = m*n_image + i;
        newval = vgrad[idx];

        // vgrad^i = Y'q1^i
        for (kk = 0; kk < l_labels; kk++) {
            // km,ki->mi
            newval += Y[kk*l_shm + m]*q1[kk*n_image + i];
        }

        if (inpaint_nloc[i]) {
            // vgrad^i += M Y'q2^i
            for (kk = 0; kk < l_labels; kk++) {
                // km,ki->mi
                newval += Y[kk*l_shm + m]*M[m]*q2[kk*n_image + i];
            }
        }

        vgrad[idx] = newval;
    }
}

#ifdef precond
__global__ void prox_primal(double *x, double *xtau)
#else
__global__ void prox_primal(double *x, double tau)
#endif
{
    /* u1[~uconstrloc] = max(0, u1)
     * u1[uconstrloc] = constraint_u[uconstrloc]
     *
     * u2 = 1/(1 + tau) max(u2 + tau*fl, min(u2 + tau*fu, (1 + tau)*u2))
     */

    SUBVAR_x_u1(u1,x)
    SUBVAR_x_u2(u2,x)

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if (i >= n_image || k >= l_labels)
       return;

    // u1[k,i]
    int idx = k*n_image + i;
    double newval;

    if (uconstrloc[i]) {
        // u[uconstrloc] = constraint_u[uconstrloc]
        newval = constraint_u[idx];
    } else {
        // ~uconstrloc
        newval = u1[idx];
        // u1 = max(0, u1)
        newval = fmax(0.0,  newval);
    }
    u1[idx] = newval;

    if (inpaint_nloc[i]) {
        // u2[k,i]
        idx = k*n_image + i;
        newval = u2[idx];
#ifdef precond
        SUBVAR_x_u2(u2tau,xtau)
        double tau = u2tau[idx];
#endif
        // u2 = 1/(1 + tau) max(u2 + tau*fl, min(u2 + tau*fu, (1 + tau)*u2))
        u2[idx] = 1.0/(1.0 + tau*b[k])*fmax(newval + tau*b[k]*fl[idx],
            fmin(newval + tau*b[k]*fu[idx], (1 + tau*b[k])*newval));
    }
}
