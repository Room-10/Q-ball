#include "hip/hip_runtime.h"

__global__ void linop1(double *x, double *ygrad)
{
    /* pgrad = D v (D is the gradient on a staggered grid)
     */

    SUBVAR_x_v(v,x)
    SUBVAR_y_p(pgrad,ygrad)

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int t = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if(k >= l_shm || i >= n_image || t >= d_image)
       return;

    // iteration variable and misc.
    int aa, base;
    double newval, fac;

    // skip points on "bottom right" boundary
    int is_boundary = false;
    int curr_i = i, curr_dim = 0;
    for(aa = d_image - 1; aa >= 0; aa--) {
        curr_dim = curr_i / skips[aa];
        curr_i = curr_i % skips[aa];
        if(curr_dim == imagedims[d_image - 1 - aa] - 1) {
            is_boundary = true;
            break;
        }
    }

    newval = 0.0;
    fac = 1.0/(double)navgskips;

    // pgrad = D v (D is the gradient on a staggered grid)
    if(!is_boundary) {
        for(aa = 0; aa < navgskips; aa++) {
            base = i + avgskips[t*navgskips + aa];
            newval +=  fac*(
                v[k*n_image + (base + skips[t])] - v[k*n_image + base]
            );
        }
    }
    pgrad[k*nd_skip + t*n_image + i] = newval;
}

__global__ void linop2(double *x, double *ygrad)
{
    /* q0grad = b'u1
     *
     * q1grad = Yv - u1
     *
     * q2grad = YMv - u2
     */

    SUBVAR_x_u1(u1,x)
    SUBVAR_x_u2(u2,x)
    SUBVAR_x_v(v,x)
    SUBVAR_y_q0(q0grad,ygrad)
    SUBVAR_y_q1(q1grad,ygrad)
    SUBVAR_y_q2(q2grad,ygrad)

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(k >= l_labels || i >= n_image)
       return;

    // iteration variables and misc.
    int mm, tt, idx;
    double newval;

    if(k == 0) {
        // q0grad = b'u1
        newval = 0.0;
        for(mm = 0; mm < l_labels; mm++) {
            newval += b[mm]*u1[mm*n_image + i];
        }
        newval *= b_precond;
        q0grad[i] = newval;
    }

    // q1grad[k,i]
    idx = k*n_image + i;
    newval = -u1[idx];

    // q1grad = Yv - u1
    for(mm = 0; mm < l_shm; mm++) {
        newval += Y[k*l_shm + mm]*v[mm*n_image + i];
    }
    q1grad[idx] = newval;

    if (inpaint_nloc[i]) {
        // q2grad[k,i]
        idx = k*n_image + i;
        newval = -u2[idx];

        // q2grad = YMv - u2
        for(mm = 0; mm < l_shm; mm++) {
            newval += Y[k*l_shm + mm]*M[mm]*v[mm*n_image + i];
        }
        q2grad[idx] = newval;
    }
}

#ifdef precond
__global__ void prox_dual(double *y, double *ysigma)
#else
__global__ void prox_dual(double *y, double sigma)
#endif
{
    /* p = proj(p, lbd)
     * q0 -= sigma
     */

    SUBVAR_y_p(p,y)
    SUBVAR_y_q0(q0,y)

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(i >= n_image || k >= l_labels)
       return;

    // iteration variables and misc.
    int mm, tt, idx;
    double norm = 0.0, newval;

    if (k == 0) {
        for(mm = 0; mm < l_shm; mm++) {
            for(tt = 0; tt < d_image; tt++) {
                idx = mm*nd_skip + tt*n_image + i;
                norm += p[idx]*p[idx];
            }
        }

        if(norm > lbd*lbd) {
            norm = lbd/sqrt(norm);
            for(mm = 0; mm < l_shm; mm++) {
                for(tt = 0; tt < d_image; tt++) {
                    p[mm*nd_skip + tt*n_image + i] *= norm;
                }
            }
        }

#ifdef precond
        SUBVAR_y_q0(q0sigma,ysigma)
        q0[i] -= q0sigma[i]*b_precond;
#else
        q0[i] -= sigma*b_precond;
#endif
    }
}
