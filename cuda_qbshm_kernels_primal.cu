#include "hip/hip_runtime.h"

__global__ void PrimalKernel1(double *uk, double *vk, double *wk, double *w0k,
                              double *ubark, double *vbark, double *wbark, double *w0bark,
                              double *pk, double *gk, double *q0k, double *q1k, double *p0k, double *g0k,
                              double *pkp1, double *gkp1, double *q0kp1, double *q1kp1, double *p0kp1, double *g0kp1,
                              double *b, double *A, double *B, long *P,
                              double *f, double *Y, double sigma, double tau, double theta,
                              double lbd, double b_precond,
                              double *constraint_u, unsigned char *uconstrloc)
{
    /* ubark = 0
     * ubark += diag(b) D' pkp1 (D' = -div with Dirichlet boundary)
     */

    // global thread index
    int k = blockIdx.x*blockDim.x + threadIdx.x;

    // stay inside maximum dimensions
    if(k >= l_labels)
       return;

    // iteration variables and misc.
    int ii, aa, tt, is_boundary, curr_dim, curr_i, base;

    for(ii = 0; ii < n_image; ii++) {
        ubark[k*n_image + ii] = 0.0;
    }

    for(ii = 0; ii < n_image; ii++) {
        // skip points on "bottom right" boundary
        is_boundary = false; curr_i = ii;
        for(tt = d_image - 1; tt >= 0; tt--) {
            curr_dim = curr_i / skips[tt];
            curr_i = curr_i % skips[tt];
            if(curr_dim == imagedims[d_image - 1 - tt] - 1) {
                is_boundary = true;
                break;
            }
        }

        if(!is_boundary) {
            for(tt = 0; tt < d_image; tt++) {
                for(aa = 0; aa < navgskips; aa++) {
                    base = ii + avgskips[tt*navgskips + aa];
                    ubark[k*n_image + (base + skips[tt])] +=
                        b[k]/(double)navgskips * pkp1[k*nd_skip + tt*n_image + ii];
                    ubark[k*n_image + base] -=
                        b[k]/(double)navgskips * pkp1[k*nd_skip + tt*n_image + ii];
                }
            }
        }
    }
}

__global__ void PrimalKernel2(double *uk, double *vk, double *wk, double *w0k,
                              double *ubark, double *vbark, double *wbark, double *w0bark,
                              double *pk, double *gk, double *q0k, double *q1k, double *p0k, double *g0k,
                              double *pkp1, double *gkp1, double *q0kp1, double *q1kp1, double *p0kp1, double *g0kp1,
                              double *b, double *A, double *B, long *P,
                              double *f, double *Y, double sigma, double tau, double theta,
                              double lbd, double b_precond,
                              double *constraint_u, unsigned char *uconstrloc)
{
    /* wbark^ij = A^j gkp1_t^ij - B^j P^j pkp1_t^i
     * wbark = wk - tau*wbark
     * wbark, wk = wbark + theta*(wbark - wk), wbark
     *
     * (W1:)
     * w0bark^ij = A^j g0kp1_t^ij - B^j P^j p0kp1_t^i
     * w0bark = w0k - tau*w0bark
     * w0bark, w0k = w0bark + theta*(w0bark - w0k), w0bark
     */

    // global thread index
    int _lj = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int t = blockIdx.z*blockDim.z + threadIdx.z;

    // stay inside maximum dimensions
    if(_lj >= sm_skip || i >= n_image || t >= d_image)
       return;

    // these have to be computed on the fly
    int l = _lj / m_gradients;
    int j = _lj % m_gradients;

    // iteration variable and misc.
    int mm;
    double wbark_tmp;

    wbark_tmp = 0.0;
    for(mm = 0; mm < s_manifold; mm++) {
        // jlm,ijmt->ijlt
        wbark_tmp += A[j*ss_skip + l*s_manifold + mm] *
                    gkp1[i*msd_skip + j*sd_skip + mm*d_image + t];
    }

    for(mm = 0; mm < r_points; mm++) {
        // jlm,jmti->ijlt
        wbark_tmp -= B[j*sr_skip + l*r_points + mm] *
                    pkp1[P[j*r_points + mm]*nd_skip + t*n_image + i];
    }

    wbark_tmp = wk[i*msd_skip + j*sd_skip + l*d_image + t] - tau*wbark_tmp;

    wbark[i*msd_skip + j*sd_skip + l*d_image + t] = wbark_tmp
        + theta*(wbark_tmp - wk[i*msd_skip + j*sd_skip + l*d_image + t]);
    wk[i*msd_skip + j*sd_skip + l*d_image + t] = wbark_tmp;

#if 'W' == dataterm
    if(t == 0) {
        wbark_tmp = 0.0;
        for(mm = 0; mm < s_manifold; mm++) {
            // jlm,ijm->ijl
            wbark_tmp += A[j*ss_skip + l*s_manifold + mm] *
                        g0kp1[i*sm_skip + j*s_manifold + mm];
        }
        for(mm = 0; mm < r_points; mm++) {
            // jlm,jmi->ijl
            wbark_tmp -= B[j*sr_skip + l*r_points + mm] *
                        p0kp1[P[j*r_points + mm]*n_image + i];
        }
        wbark_tmp = w0k[i*sm_skip + j*s_manifold + l] - tau*wbark_tmp;

        w0bark[i*sm_skip + j*s_manifold + l] = wbark_tmp
            + theta*(wbark_tmp - w0k[i*sm_skip + j*s_manifold + l]);
        w0k[i*sm_skip + j*s_manifold + l] = wbark_tmp;
    }
#endif
}

__global__ void PrimalKernel3(double *uk, double *vk, double *wk, double *w0k,
                              double *ubark, double *vbark, double *wbark, double *w0bark,
                              double *pk, double *gk, double *q0k, double *q1k, double *p0k, double *g0k,
                              double *pkp1, double *gkp1, double *q0kp1, double *q1kp1, double *p0kp1, double *g0kp1,
                              double *b, double *A, double *B, long *P,
                              double *f, double *Y, double sigma, double tau, double theta,
                              double lbd, double b_precond,
                              double *constraint_u, unsigned char *uconstrloc)
{
    /* ubark += b q0kp1' - q1kp1 - diag(b) f (quadratic)
     * ubark += b q0kp1' - q1kp1 + diag(b) p0kp1 (W1)
     * ubark = dataterm_factor*(uk - tau*ubark)
     * ubark[~uconstrloc] = max(0, ubark)
     * ubark[uconstrloc] = constraint_u[uconstrloc]
     * ubark, uk = ubark + theta*(ubark - uk), ubark
     */

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(i >= n_image || k >= l_labels)
       return;

    // misc.
    double ubark_tmp;

    double dataterm_factor = 1.0;
#if 'Q' == dataterm
    dataterm_factor = 1.0/(1.0 + tau*b[k]);
#endif

    if(uconstrloc[i]) {
        ubark_tmp = constraint_u[k*n_image + i];
    } else {
        ubark_tmp = ubark[k*n_image + i];
        ubark_tmp += b[k]*(b_precond*q0kp1[i]);
        ubark_tmp -= q1kp1[k*n_image + i];
#if 'Q' == dataterm
        ubark_tmp -= b[k]*f[k*n_image + i];
#elif 'W' == dataterm
        ubark_tmp += b[k]*p0kp1[k*n_image + i];
#endif
        ubark_tmp = dataterm_factor*(uk[k*n_image + i] - tau*ubark_tmp);
        ubark_tmp = fmax(0.0,  ubark_tmp);
    }

    ubark[k*n_image + i] = ubark_tmp + theta*(ubark_tmp - uk[k*n_image + i]);
    uk[k*n_image + i] = ubark_tmp;
}

__global__ void PrimalKernel4(double *uk, double *vk, double *wk, double *w0k,
                              double *ubark, double *vbark, double *wbark, double *w0bark,
                              double *pk, double *gk, double *q0k, double *q1k, double *p0k, double *g0k,
                              double *pkp1, double *gkp1, double *q0kp1, double *q1kp1, double *p0kp1, double *g0kp1,
                              double *b, double *A, double *B, long *P,
                              double *f, double *Y, double sigma, double tau, double theta,
                              double lbd, double b_precond,
                              double *constraint_u, unsigned char *uconstrloc)
{
    /* vbark^i = Y' q1kp1^i
     * vbark = vk - tau*vbark
     * vbark, vk = vbark + theta*(vbark - vk), vbark
     */

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int m = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if(i >= n_image || m >= l_shm)
       return;

    // iteration variable and misc.
    int k;
    double vbark_tmp;

    vbark_tmp = 0.0;
    for(k = 0; k < l_labels; k++) {
        // km,ki->mi
        vbark_tmp += Y[k*l_shm + m]*q1kp1[k*n_image + i];
    }

    vbark_tmp = vk[m*n_image + i] - tau*vbark_tmp;

    vbark[m*n_image + i] = vbark_tmp + theta*(vbark_tmp - vk[m*n_image + i]);
    vk[m*n_image + i] = vbark_tmp;
}
